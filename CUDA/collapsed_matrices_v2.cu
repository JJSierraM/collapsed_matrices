#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include <math.h>

typedef unsigned int uint;

inline uint nCr (uint n, uint r){
    uint output = 1;
    if (n<r) {output = 0;}
    else {
        uint i;
        for (i=1; i<r+1; i++) {output=(output*(i+n-r))/i;}
    }
    return output;
}

typedef struct Body {
    float mass;
    float2 position;
    float2 speed;
    float2 accel; 
} Body;

void body_initialize(Body* body) {
    body->mass=0.001;
    body->position = float2();
    body->speed = float2();
    body->accel = float2();
}

//Random mass and position, speed and accel set to 0
void body_random(Body* body) {
    const float rand_mass = ((float)rand()/(float)RAND_MAX)*(1.0-0.001)+0.001; 
    float2 rand_pos;
    vector2D_random(&rand_pos, -1.0, 1.0); 
    body->mass = rand_mass;
    body->position = rand_pos;
    vector2D_initialize(&body->speed);
    vector2D_initialize(&body->accel);
}

Body* body_new() {
    Body* output = (Body*) malloc(sizeof(Body));
    body_initialize(output);
    return output;
}

void body_print(Body* body) {
    printf("Mass:\t\t%06.3f\n", body->mass);
    printf("Position:\tx: %06.3f, y: %06.3f\n", body->position.x, body->position.y);
    printf("Speed:\t\tx: %06.3f, y: %06.3f\n", body->speed.x, body->speed.y);
    printf("Acceleration:\tx: %06.3f, y: %06.3f\n", body->accel.x, body->accel.y);
}

typedef struct CollapsedMatrix 
{
    uint dimension;         // Number of items the relations involves   (max 65,535)
    uint n;                 // Number of items to relate                (max 65,535) 
    uint length;            // Number of total relations                (max 65,535)

    uint* indices;          // Vector of indices, contiguous. Length=dimension*length
    Vector2D* results;      // Vector of resulted calculations
    Vector2D* sum;          // Vector of sum of the results by index 
} CollapsedMatrix;

inline void transfer_array(uint* arr1, uint* arr2, uint dim_arr2){
    for (int i=0; i<dim_arr2; i++){
        *(arr1+i)=*(arr2+i);
    }
}

void set_indices(CollapsedMatrix* collapsed_matrix) {
    uint dimension = collapsed_matrix->dimension;
    uint length = collapsed_matrix->length;

    uint *item = (uint*) malloc(dimension*sizeof(uint));
    uint i,j,k;
    for (i=0; i<dimension; i++) item[i] = dimension -1 -i;
    transfer_array(collapsed_matrix->indices+0, item, dimension);
    for (i=1; i<length; i++){
        for (j=dimension-1; j>=0; j--){
            if (j == 0)
            {
                item[0]++;
                for (k=j+1; k<dimension; k++){
                    item[k] = dimension -1 -k;
                }
                break;
            }
            else if (item[j]+1 != item[j-1])
            {
                item[j]++;
                for (k=j+1; k<dimension; k++){
                    item[k] = dimension -1 -k;
                }
                break;
            }
        }
        transfer_array(collapsed_matrix->indices+(i*dimension), item, dimension);
    }
    free(item);
}

void initialize_results(CollapsedMatrix* collapsed_matrix) {
    for (int i=0; i<collapsed_matrix->length; i++) vector2D_initialize(collapsed_matrix->results+i);
}

void initialize_sum(CollapsedMatrix* collapsed_matrix) {
    for (int i=0; i<collapsed_matrix->n; i++) vector2D_initialize(collapsed_matrix->sum+i);
}

CollapsedMatrix* collapsed_matrix_new(uint dimension_, uint n_) {
    CollapsedMatrix* collapsed_matrix = (CollapsedMatrix*) malloc(sizeof(CollapsedMatrix));
    collapsed_matrix->dimension = dimension_;
    collapsed_matrix->n = n_;

    collapsed_matrix->length = nCr(n_, dimension_);
    collapsed_matrix->indices = (uint*) malloc(collapsed_matrix->length*dimension_*sizeof(uint));
    collapsed_matrix->results = (Vector2D*) malloc(collapsed_matrix->length*sizeof(Vector2D));
    collapsed_matrix->sum = (Vector2D*) malloc(collapsed_matrix->n*sizeof(Vector2D));

    set_indices(collapsed_matrix);

    initialize_results(collapsed_matrix);
    initialize_sum(collapsed_matrix);

    return collapsed_matrix;
}

void collapsed_matrix_destroy(CollapsedMatrix* collapsed_matrix) {
    free(collapsed_matrix->indices);
    free(collapsed_matrix->results);
    free(collapsed_matrix->sum);
    free(collapsed_matrix);
}

static Vector2D grav_force(const Body* bodies) {
    const float GRAV_CONST = 5.0;
    const Vector2D vec12 = vector2D_subtract(bodies[0].position, bodies[1].position);
    const float force_mag = -GRAV_CONST * (bodies[0].mass*bodies[1].mass)/vector2D_sqr_length(vec12);
    const float force_angle = vector2D_angle(vec12);
    return vector2D_from_magnitude_and_angle(force_mag, force_angle);
}

static Vector2D grav_force_2(const Body* body_A, const Body* body_B) {
    const float GRAV_CONST = 5.0;
    const Vector2D vec12 = vector2D_subtract(body_A->position, body_B->position);
    const float force_mag = -GRAV_CONST * (body_A->mass*body_B->mass)/vector2D_sqr_length(vec12);
    const float force_angle = vector2D_angle(vec12);
    return vector2D_from_magnitude_and_angle(force_mag, force_angle);
}

void collapsed_matrix_apply_function(CollapsedMatrix* collapsed_matrix, const Body* items) {
    uint i;
    #pragma omp parallel private (i) 
    {
        #pragma omp for
        for (i=0; i<collapsed_matrix->length*collapsed_matrix->dimension; i+=collapsed_matrix->dimension){
            collapsed_matrix->results[i/collapsed_matrix->dimension] = grav_force_2(items+collapsed_matrix->indices[i], items+collapsed_matrix->indices[i+1]);
        }
    }
}

void collapsed_matrix_calculate_sum(CollapsedMatrix* collapsed_matrix) {
    uint i;
    # pragma omp parallel private ( i )
    {
        # pragma omp for
        for (i = 0; i < (collapsed_matrix->length*collapsed_matrix->dimension); i++){
            vector2D_add_equals(&collapsed_matrix->sum[collapsed_matrix->indices[i]], &collapsed_matrix->results[i/collapsed_matrix->dimension]);
        }
    }
}

struct cuBody {
    float mass;
    float2 position;
    float2 speed;
    float2 accel;
};

static __device__ __inline__ float float2_sqr_length(float2 vec) {
    return vec.x*vec.x+vec.y*vec.y;
}

static __device__ __inline__ float float2_angle(float2 vector) {
    return atan2(vector.y, vector.x);
}

static __device__ __inline__ float2 float2_from_magnitude_and_angle(float magnitude, float angle) {
    float2 output;
    output.x = magnitude*cos(angle);
    output.y = magnitude*sin(angle);
    return output;
}

static __device__ __inline__ float2 float2_subtract(float2 A, float2 B) {
    return make_float2(A.x-B.x, A.y-B.y);
}

static __device__ __inline__ void float2_add_equals(float2* a, const float2* b) {
    a->x = a->x + b->x;
    a->y = a->y + b->y;
}

struct cuCollapsedMatrix 
{
    uint dimension;         // Number of items the relations involves   (max 65,535)
    uint n;                 // Number of items to relate                (max 65,535) 
    uint length;            // Number of total relations                (max 65,535)

    uint* indices;          // Vector of indices, contiguous. Length=dimension*length
    float2* results;        // Vector of resulted calculations
    float2* sum;            // Vector of sum of the results by index 
};

__device__ uint cu_nCr(uint n, uint r) {
    uint output = 1;
    if (n<r) {output = 0;}
    else {
        uint i;
        for (i=1; i<r+1; i++) {output=(output*(i+n-r))/i;}
    }
    return output;
}

//grid(1,1,1)
__global__ void cu_collapsed_matrix_new(cuCollapsedMatrix* cu_collapsed_matrix, uint dimension_, uint n_) {
    cu_collapsed_matrix = (cuCollapsedMatrix*) malloc(sizeof(cuCollapsedMatrix));
    cu_collapsed_matrix->dimension = dimension_;
    cu_collapsed_matrix->n = n_;

    cu_collapsed_matrix->length = cu_nCr(n_, dimension_);
    cu_collapsed_matrix->indices = (uint*) malloc(cu_collapsed_matrix->length*dimension_*sizeof(uint));
    cu_collapsed_matrix->results = (float2*) malloc(cu_collapsed_matrix->length*sizeof(float2));
    cu_collapsed_matrix->sum = (float2*) malloc(cu_collapsed_matrix->n*sizeof(float2));
}

__device__ void cu_set_indices(uint* indices, uint dimension, int x) {
    uint prev = 0;
    uint* item = indices+(x*dimension);
    for (uint k=0; k<dimension; k++){
        for (uint j=0; j<999; j++){
            if ((x-prev) < cu_nCr(dimension-k+j, j)) {
                item[k] = j + dimension -k -1;
                prev += cu_nCr(dimension -k +j -1, j-1);
                break;
            }
        }
    }
}

__device__ void cu_initialize_results(uint length, float2* results, int x) {
    if (x<length) {
    results[x] = float2();
    }
}

__device__ void cu_initialize_sum(uint n, float2* sum, int x) {
    if (x<n) {
    sum[x] = float2();
    }
}

__global__ void cu_collapsed_matrix_initialize(uint dimension, uint n, uint* indices, float2* results, float2* sum) {
    int x = blockIdx.x * blockDim.x+ threadIdx.x;
    cu_set_indices(indices, dimension, x);
    // cu_initialize_results(cu_nCr(n,dimension), results, x);
    // cu_initialize_sum(n, sum, x);
}

__device__ float2 dev_grav_force(cuBody bodyA, cuBody bodyB) {
    const float GRAV_CONST = 5.0;
    const float2 vec12 = float2_subtract(bodyA.position, bodyB.position);
    return make_float2(bodyA.position.x, bodyA.position.y);
    const float force_mag = -GRAV_CONST * (bodyA.mass*bodyB.mass)/float2_sqr_length(vec12);
    const float force_angle = float2_angle(vec12);
    return float2_from_magnitude_and_angle(force_mag, force_angle);
}

__global__ void calculate_gravity (const uint* indices, float2* results, const cuBody* bodies) {
    int x = blockIdx.x * blockDim.x+ threadIdx.x;
    if (x==0) {results[x] = bodies[1].position; return;}
    results[x] = dev_grav_force(bodies[indices[2*x]], bodies[indices[2*x+1]]);
}

//grid(length*dimension)
__global__ void calculate_sum (uint length, uint dimension, uint* indices, float2* results, float2* sum) {
    int x = blockIdx.x * blockDim.x+ threadIdx.x;
    for (uint i=0; i< (length*dimension); i++) {
    if (x == indices[i]) float2_add_equals(&sum[indices[i]], &results[i/dimension]);
    }
}

void cu_collapsed_matrix_new (CollapsedMatrix* from, cuCollapsedMatrix* to) {
    to->dimension = from->dimension;
    to->n = from->n;
    to->length = nCr(to->n, to->dimension);
    hipMalloc((void**) &to->indices, from->length*from->dimension*sizeof(uint));
    hipMalloc((void**) &to->results, from->length*sizeof(float2));
    hipMalloc((void**) &to->sum, from->n*sizeof(float2));

    hipMemset(&to->results, 0, from->length*sizeof(float2));
    hipMemset(&to->sum, 0, from->n*sizeof(float2));
    // hipMemcpy(to->indices, from->indices, from->length*from->dimension*sizeof(uint), hipMemcpyHostToDevice);
    // hipMemcpy(to->results, from->results, from->length*sizeof(float2), hipMemcpyHostToDevice);
    // hipMemcpy(to->sum, from->sum, from->n*sizeof(float2), hipMemcpyHostToDevice);
}

int main() {
    uint n=3, d=2;
    srand(3);

    clock_t start_fun, end_fun, start_sum, end_sum;

    Body* bodies = (Body*) malloc(n*sizeof(Body));
    for (int i=0; i<n; i++) {body_random(bodies+i);}

    CollapsedMatrix* collapsed_matrix = collapsed_matrix_new(d, n);
    float2* sum = (float2*) malloc(n*sizeof(float2));
    uint* indices = (uint*) malloc(nCr(n,d)*sizeof(uint));
    ////////////////////////////
    cuCollapsedMatrix* dev_CM = (cuCollapsedMatrix*) malloc(sizeof(cuCollapsedMatrix));;
    cu_collapsed_matrix_new(collapsed_matrix, dev_CM);
    cuBody* dev_bodies;
    Body test_body[1];
    hipMalloc((void**) &dev_bodies, n*sizeof(cuBody));
    hipMemcpy(dev_bodies, bodies, n*sizeof(cuBody), hipMemcpyHostToDevice);
    
    collapsed_matrix_apply_function(collapsed_matrix, bodies);
    collapsed_matrix_calculate_sum(collapsed_matrix);
    
    dim3 block(1);
    dim3 grid(1);
    // cu_collapsed_matrix_new<<<grid,1>>>(dev_CM, d, n);
    printf("Matrix created\n");
    block = dim3(dev_CM->n);
    grid = dim3(dev_CM->length/(dev_CM->n-1));
    printf("Initializing\n");
    cu_collapsed_matrix_initialize<<<grid,block>>>(dev_CM->dimension, dev_CM->n, dev_CM->indices, dev_CM->results, dev_CM->sum);
    printf("initialized\n");
    ////////////////////////////
    start_fun = clock();
    calculate_gravity<<<grid,block>>> (dev_CM->indices, dev_CM->results, dev_bodies);
    printf("Gravity calculated\n");
    block = dim3((int)sqrtf(n));
    grid = dim3(n/block.x);
    calculate_sum<<<grid,block>>> (dev_CM->length, dev_CM->dimension, dev_CM->indices, dev_CM->results, dev_CM->sum);
    printf("Gravity added\n");

    // hipMemcpy(results, dev_CM->sum, collapsed_matrix->n*sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(sum, dev_CM->results, n*sizeof(float2), hipMemcpyDeviceToHost);

    printf("Results transfered\n");
    // for (int i=0; i<nCr(n,d); i++) {grav_force(bodies);}
    end_fun = clock();

    start_sum = clock();
    //collapsed_matrix_calculate_sum(collapsed_matrix);
    end_sum = clock();

    // start_fun = clock();
    // Vector2D force;
    // for (int i = 1; i < n; i++)
    // {
    //     for (int j = 0; j < i; j++)
    //     {
    //         force = grav_force_2(bodies+i, bodies+j);
    //         vector2D_add_equals(collapsed_matrix->results, &force);
    //     }
    // }
    // end_fun = clock();

    printf("\nTime spent:\n");
    printf("Apply function: %f\n", ((double) (end_fun-start_fun)) / CLOCKS_PER_SEC * 1000);
    printf("Apply sumation: %f\n", ((double) (end_sum-start_sum)) / CLOCKS_PER_SEC * 1000);
    
    body_print(&bodies[0]);
    body_print(&bodies[1]);
    printf("Sum[0] = \t");
    printf("(%06.3f,%06.3f)\n", collapsed_matrix->results[0].x, collapsed_matrix->results[0].y);
    printf("Dev_Sum[0] = \t");
    printf("(%06.3f,%06.3f)\n", sum[0].x, sum[0].y);

    return 0;
}